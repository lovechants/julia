
#include <hip/hip_runtime.h>
__global__ void add_vectors(const float *a, float *b, float *c, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    c[idx] = a[idx] + b[idx];
  }
}
